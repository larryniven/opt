#include "hip/hip_runtime.h"
#include "opt/opt-gpu.h"
#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <hip/hip_runtime.h>

namespace opt {

    namespace gpu {

        struct iadagrad_update_op {
            double step_size;

            template <class T>
            __host__ __device__
            void operator()(T t) const
            {
                auto& theta = thrust::get<0>(t);
                auto& loss_grad = thrust::get<1>(t);
                auto& accu_grad_sq = thrust::get<2>(t);

                accu_grad_sq += pow(loss_grad, 2);

                if (accu_grad_sq > 0) {
                    theta -= loss_grad * step_size
                        / sqrt(accu_grad_sq);
                }
            }
        };

        void adagrad_update(la::gpu::vector_like<double>& theta,
            la::gpu::vector_like<double> const& loss_grad,
            la::gpu::vector_like<double>& accu_grad_sq,
            double step_size)
        {
            thrust::for_each(
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.begin()),
                    thrust::device_ptr<double const>(loss_grad.begin()),
                    thrust::device_ptr<double>(accu_grad_sq.begin()))),
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.end()),
                    thrust::device_ptr<double const>(loss_grad.end()),
                    thrust::device_ptr<double>(accu_grad_sq.end()))),
                iadagrad_update_op { step_size });
        }

        void adagrad_update(la::gpu::matrix_like<double>& theta,
            la::gpu::matrix_like<double> const& loss_grad,
            la::gpu::matrix_like<double>& accu_grad_sq,
            double step_size)
        {
            unsigned int size = theta.rows() * theta.cols();

            thrust::for_each(
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.data()),
                    thrust::device_ptr<double const>(loss_grad.data()),
                    thrust::device_ptr<double>(accu_grad_sq.data()))),
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.data() + size),
                    thrust::device_ptr<double const>(loss_grad.data() + size),
                    thrust::device_ptr<double>(accu_grad_sq.data() + size))),
                iadagrad_update_op { step_size });
        }

        struct iadam_update_op {
            double time;
            double alpha;
            double beta1;
            double beta2;
            double b1;
            double b2;

            template <class T>
            __host__ __device__
            void operator()(T t) const
            {
                auto& theta = thrust::get<0>(t);
                auto& loss_grad = thrust::get<1>(t);
                auto& first_moment = thrust::get<2>(t);
                auto& second_moment = thrust::get<3>(t);

                first_moment = first_moment * beta1 + loss_grad * (1 - beta1);
                second_moment = second_moment * beta2 + pow(loss_grad, 2) * (1 - beta2);

                theta -= alpha * first_moment / b1
                    / (std::sqrt(second_moment / b2) + 1e-8);
            }
        };

        void adam_update(la::gpu::vector_like<double>& theta,
            la::gpu::vector_like<double> const& loss_grad,
            la::gpu::vector_like<double>& first_moment,
            la::gpu::vector_like<double>& second_moment,
            double time, double alpha, double beta1, double beta2)
        {
            thrust::for_each(
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.begin()),
                    thrust::device_ptr<double const>(loss_grad.begin()),
                    thrust::device_ptr<double>(first_moment.begin()),
                    thrust::device_ptr<double>(second_moment.begin()))),
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.end()),
                    thrust::device_ptr<double const>(loss_grad.end()),
                    thrust::device_ptr<double>(first_moment.end()),
                    thrust::device_ptr<double>(second_moment.end()))),
                iadam_update_op { time, alpha, beta1, beta2, 1 - pow(beta1, time), 1 - pow(beta2, time) });
        }

        void adam_update(la::gpu::matrix_like<double>& theta,
            la::gpu::matrix_like<double> const& loss_grad,
            la::gpu::matrix_like<double>& first_moment,
            la::gpu::matrix_like<double>& second_moment,
            double time, double alpha, double beta1, double beta2)
        {
            unsigned int size = theta.rows() * theta.cols();

            thrust::for_each(
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.data()),
                    thrust::device_ptr<double const>(loss_grad.data()),
                    thrust::device_ptr<double>(first_moment.data()),
                    thrust::device_ptr<double>(second_moment.data()))),
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.data() + size),
                    thrust::device_ptr<double const>(loss_grad.data() + size),
                    thrust::device_ptr<double>(first_moment.data() + size),
                    thrust::device_ptr<double>(second_moment.data() + size))),
                iadam_update_op { time, alpha, beta1, beta2, 1 - pow(beta1, time), 1 - pow(beta2, time) });
        }

    }
}

