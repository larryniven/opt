#include "hip/hip_runtime.h"
#include "opt/opt-gpu.h"
#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <hip/hip_runtime.h>

namespace opt {

    namespace gpu {

        struct iconst_step_update_op {
            double step_size;

            template <class T>
            __host__ __device__
            void operator()(T t) const
            {
                auto& theta = thrust::get<0>(t);
                auto& loss_grad = thrust::get<1>(t);

                theta -= step_size * loss_grad;
            }
        };

        void const_step_update(la::gpu::vector_like<double>& theta,
            la::gpu::vector_like<double> const& loss_grad,
            double step_size)
        {
            thrust::for_each(
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.begin()),
                    thrust::device_ptr<double const>(loss_grad.begin()))),
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.end()),
                    thrust::device_ptr<double const>(loss_grad.end()))),
                iconst_step_update_op { step_size });
        }

        void const_step_update(la::gpu::matrix_like<double>& theta,
            la::gpu::matrix_like<double> const& loss_grad,
            double step_size)
        {
            const_step_update(theta.as_vector(), loss_grad.as_vector(), step_size);
        }

        void const_step_update(la::gpu::tensor_like<double>& theta,
            la::gpu::tensor_like<double> const& loss_grad,
            double step_size)
        {
            const_step_update(theta.as_vector(), loss_grad.as_vector(), step_size);
        }

        struct iconst_step_update_momentum_op {
            double momentum;
            double step_size;

            template <class T>
            __host__ __device__
            void operator()(T t) const
            {
                auto& theta = thrust::get<0>(t);
                auto& loss_grad = thrust::get<1>(t);
                auto& update = thrust::get<2>(t);

                update = update * momentum + loss_grad * (1 - momentum);
                theta -= step_size * update;
            }
        };

        void const_step_update_momentum(la::gpu::vector_like<double>& theta,
            la::gpu::vector_like<double> const& loss_grad,
            la::gpu::vector_like<double>& update,
            double momentum,
            double step_size)
        {
            thrust::for_each(
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.begin()),
                    thrust::device_ptr<double const>(loss_grad.begin()),
                    thrust::device_ptr<double>(update.begin()))),
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.end()),
                    thrust::device_ptr<double const>(loss_grad.end()),
                    thrust::device_ptr<double>(update.end()))),
                iconst_step_update_momentum_op { momentum, step_size });
        }

        void const_step_update_momentum(la::gpu::matrix_like<double>& theta,
            la::gpu::matrix_like<double> const& loss_grad,
            la::gpu::matrix_like<double>& update,
            double momentum,
            double step_size)
        {
            const_step_update_momentum(theta.as_vector(), loss_grad.as_vector(),
                update.as_vector(), momentum, step_size);
        }

        void const_step_update_momentum(la::gpu::tensor_like<double>& theta,
            la::gpu::tensor_like<double> const& loss_grad,
            la::gpu::tensor_like<double>& update,
            double momentum,
            double step_size)
        {
            const_step_update_momentum(theta.as_vector(), loss_grad.as_vector(),
                update.as_vector(), momentum, step_size);
        }

        struct iadagrad_update_op {
            double step_size;

            template <class T>
            __host__ __device__
            void operator()(T t) const
            {
                auto& theta = thrust::get<0>(t);
                auto& loss_grad = thrust::get<1>(t);
                auto& accu_grad_sq = thrust::get<2>(t);

                accu_grad_sq += pow(loss_grad, 2);

                if (accu_grad_sq > 0) {
                    theta -= loss_grad * step_size
                        / sqrt(accu_grad_sq);
                }
            }
        };

        void adagrad_update(la::gpu::vector_like<double>& theta,
            la::gpu::vector_like<double> const& loss_grad,
            la::gpu::vector_like<double>& accu_grad_sq,
            double step_size)
        {
            thrust::for_each(
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.begin()),
                    thrust::device_ptr<double const>(loss_grad.begin()),
                    thrust::device_ptr<double>(accu_grad_sq.begin()))),
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.end()),
                    thrust::device_ptr<double const>(loss_grad.end()),
                    thrust::device_ptr<double>(accu_grad_sq.end()))),
                iadagrad_update_op { step_size });
        }

        void adagrad_update(la::gpu::matrix_like<double>& theta,
            la::gpu::matrix_like<double> const& loss_grad,
            la::gpu::matrix_like<double>& accu_grad_sq,
            double step_size)
        {
            adagrad_update(theta.as_vector(), loss_grad.as_vector(),
                accu_grad_sq.as_vector(), step_size);
        }

        void adagrad_update(la::gpu::tensor_like<double>& theta,
            la::gpu::tensor_like<double> const& loss_grad,
            la::gpu::tensor_like<double>& accu_grad_sq,
            double step_size)
        {
            adagrad_update(theta.as_vector(), loss_grad.as_vector(),
                accu_grad_sq.as_vector(), step_size);
        }

        struct irmsprop_update_op {
            double decay;
            double step_size;

            template <class T>
            __host__ __device__
            void operator()(T t) const
            {
                auto& theta = thrust::get<0>(t);
                auto& loss_grad = thrust::get<1>(t);
                auto& accu_grad_sq = thrust::get<2>(t);

                accu_grad_sq = decay * accu_grad_sq
                    + (1 - decay) * std::pow(loss_grad, 2);

                if (accu_grad_sq > 0) {
                    theta -= loss_grad * step_size
                        / std::sqrt(accu_grad_sq);
                }
            }
        };

        void rmsprop_update(la::gpu::vector_like<double>& theta,
            la::gpu::vector_like<double> const& loss_grad,
            la::gpu::vector_like<double>& accu_grad_sq,
            double decay,
            double step_size)
        {
            thrust::for_each(
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.begin()),
                    thrust::device_ptr<double const>(loss_grad.begin()),
                    thrust::device_ptr<double>(accu_grad_sq.begin()))),
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.end()),
                    thrust::device_ptr<double const>(loss_grad.end()),
                    thrust::device_ptr<double>(accu_grad_sq.end()))),
                irmsprop_update_op { decay, step_size });
        }

        void rmsprop_update(la::gpu::matrix_like<double>& theta,
            la::gpu::matrix_like<double> const& loss_grad,
            la::gpu::matrix_like<double>& accu_grad_sq,
            double decay,
            double step_size)
        {
            rmsprop_update(theta.as_vector(), loss_grad.as_vector(),
                accu_grad_sq.as_vector(), decay, step_size);
        }

        void rmsprop_update(la::gpu::tensor_like<double>& theta,
            la::gpu::tensor_like<double> const& loss_grad,
            la::gpu::tensor_like<double>& accu_grad_sq,
            double decay,
            double step_size)
        {
            rmsprop_update(theta.as_vector(), loss_grad.as_vector(),
                accu_grad_sq.as_vector(), decay, step_size);
        }

        struct iadam_update_op {
            int time;
            double alpha;
            double beta1;
            double beta2;
            double b1;
            double b2;

            template <class T>
            __host__ __device__
            void operator()(T t) const
            {
                auto& theta = thrust::get<0>(t);
                auto& loss_grad = thrust::get<1>(t);
                auto& first_moment = thrust::get<2>(t);
                auto& second_moment = thrust::get<3>(t);

                first_moment = first_moment * beta1 + loss_grad * (1 - beta1);
                second_moment = second_moment * beta2 + pow(loss_grad, 2) * (1 - beta2);

                theta -= alpha * first_moment / b1
                    / (std::sqrt(second_moment / b2) + 1e-8);
            }
        };

        void adam_update(la::gpu::vector_like<double>& theta,
            la::gpu::vector_like<double> const& loss_grad,
            la::gpu::vector_like<double>& first_moment,
            la::gpu::vector_like<double>& second_moment,
            int& time, double alpha, double beta1, double beta2)
        {
            thrust::for_each(
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.begin()),
                    thrust::device_ptr<double const>(loss_grad.begin()),
                    thrust::device_ptr<double>(first_moment.begin()),
                    thrust::device_ptr<double>(second_moment.begin()))),
                thrust::make_zip_iterator(thrust::make_tuple(
                    thrust::device_ptr<double>(theta.end()),
                    thrust::device_ptr<double const>(loss_grad.end()),
                    thrust::device_ptr<double>(first_moment.end()),
                    thrust::device_ptr<double>(second_moment.end()))),
                iadam_update_op { time, alpha, beta1, beta2, 1 - pow(beta1, time),
                    1 - pow(beta2, time) });

            ++time;
        }

        void adam_update(la::gpu::matrix_like<double>& theta,
            la::gpu::matrix_like<double> const& loss_grad,
            la::gpu::matrix_like<double>& first_moment,
            la::gpu::matrix_like<double>& second_moment,
            int& time, double alpha, double beta1, double beta2)
        {
            adam_update(theta.as_vector(), loss_grad.as_vector(),
                first_moment.as_vector(), second_moment.as_vector(),
                time, alpha, beta1, beta2);
        }

        void adam_update(la::gpu::tensor_like<double>& theta,
            la::gpu::tensor_like<double> const& loss_grad,
            la::gpu::tensor_like<double>& first_moment,
            la::gpu::tensor_like<double>& second_moment,
            int& time, double alpha, double beta1, double beta2)
        {
            adam_update(theta.as_vector(), loss_grad.as_vector(),
                first_moment.as_vector(), second_moment.as_vector(),
                time, alpha, beta1, beta2);
        }

    }
}

